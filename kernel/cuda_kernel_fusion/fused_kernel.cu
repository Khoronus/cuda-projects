#include <hip/hip_runtime.h>
#include <iostream>

__global__ void fusedKernel(float* a, float* b, float* d, float* e, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float temp = a[idx] + b[idx];
        e[idx] = temp * d[idx];
    }
}

void computeFused(float* a, float* b, float* d, float* e, int N) {
    float *d_a, *d_b, *d_d, *d_e;
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_d, N * sizeof(float));
    hipMalloc((void**)&d_e, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    fusedKernel<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_d, d_e, N);
    hipDeviceSynchronize();

    hipMemcpy(e, d_e, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_d);
    hipFree(d_e);
}

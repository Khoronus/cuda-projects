#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernelA(float* a, float* b, float* c, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void kernelB(float* c, float* d, float* e, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        e[idx] = c[idx] * d[idx];
    }
}

void computeSeparate(float* a, float* b, float* d, float* e, int N) {
    float *d_a, *d_b, *d_c, *d_d, *d_e;
    hipMalloc((void**)&d_a, N * sizeof(float));
    hipMalloc((void**)&d_b, N * sizeof(float));
    hipMalloc((void**)&d_c, N * sizeof(float));
    hipMalloc((void**)&d_d, N * sizeof(float));
    hipMalloc((void**)&d_e, N * sizeof(float));

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_d, d, N * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    kernelA<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    kernelB<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_d, d_e, N);
    hipDeviceSynchronize();

    hipMemcpy(e, d_e, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_d);
    hipFree(d_e);
}
